#include "hip/hip_runtime.h"
﻿#pragma once

#include <cassert>
#include <hip/hip_runtime.h>
#include <cstdio>
#include <array>
#include <cstdlib>
#include <string>
#include <fstream>
#include <iostream>
#include <vector>
#include <hipblas.h>
#include <random>
#include <hip/hip_runtime.h>

#include "../../cud_helper.hpp"

template <
    uint32_t BLOCK_TILE_M = 128,
    uint32_t BLOCK_TILE_N = 128,
    uint32_t BLOCK_TILE_K = 128,
    uint32_t THREAD_TILE_M = 16,
    uint32_t THREAD_TILE_N = 16,
    uint32_t THREAD_TILE_K = 16,
    uint32_t THREAD_SUBTILE_M = 8,
    uint32_t THREAD_SUBTILE_N = 8,
    uint32_t THREAD_SUBTILE_K = 8
>
__global__ void sgemmSubTile(int M, int N, int K,
                             const float4* __restrict__ srcMatA,
                             const float4* __restrict__ srcMatB,
                             float4* __restrict__ dstMat) {
    constexpr int BLOCK_TILE_VEC_N = BLOCK_TILE_N / 4;
    constexpr int BLOCK_TILE_VEC_K = BLOCK_TILE_K / 4;
    constexpr int THREAD_TILE_VEC_N = THREAD_TILE_N / 4;
    constexpr int THREAD_SUBTILE_VEC_N = THREAD_SUBTILE_N / 4;
    constexpr int THREAD_SUBTILE_VEC_K = THREAD_SUBTILE_K / 4;

    // Shared memory
    __shared__ float4 sharedA[BLOCK_TILE_M][BLOCK_TILE_VEC_K];
    __shared__ float4 sharedB[BLOCK_TILE_K][BLOCK_TILE_VEC_N];

    // Thread indices
    const int tidx = threadIdx.x;
    const int tidy = threadIdx.y;
    const int bidx = blockIdx.x;
    const int bidy = blockIdx.y;
    const int localIndex = tidy * blockDim.x + tidx;
    const int groupThreadCount = blockDim.x * blockDim.y;

    // Accumulator registers
    float4 regAccumulator[THREAD_TILE_M][THREAD_TILE_VEC_N];

    // Zero-fill accumulator
#pragma unroll
    for (int tm = 0; tm < THREAD_TILE_M; tm++) {
#pragma unroll
        for (int tn = 0; tn < THREAD_TILE_VEC_N; tn++) {
            regAccumulator[tm][tn] = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
        }
    }

    // Helper lambda for loading global to shared memory
    auto loadGlobalToShared = [&](int globalCoordX, int globalCoordY,
                                  int globalExtentX, int globalExtentY,
                                  int globalRowStride, bool loadA) {
        const int loadsPerThread = (globalExtentX * globalExtentY) / groupThreadCount;

        for (int i = 0; i < loadsPerThread; i++) {
            const int linearIdx = i * groupThreadCount + localIndex;
            const int srcOffsetX = linearIdx % globalExtentX;
            const int srcOffsetY = linearIdx / globalExtentX;
            const int srcCoordX = globalCoordX + srcOffsetX;
            const int srcCoordY = globalCoordY + srcOffsetY;
            const int srcIndex = srcCoordY * globalRowStride + srcCoordX;

            if (loadA) {
                sharedA[srcOffsetY][srcOffsetX] = srcMatA[srcIndex];
            } else {
                sharedB[srcOffsetY][srcOffsetX] = srcMatB[srcIndex];
            }
        }
    };

    // Main loop over K dimension
    const int blockBaseM = bidy * BLOCK_TILE_M;
    const int blockBaseVecN = bidx * BLOCK_TILE_VEC_N;
    const int blockSplitKCount = K / BLOCK_TILE_K;

    for (int iBlockK = 0; iBlockK < blockSplitKCount; iBlockK++) {
        const int blockBaseK = iBlockK * BLOCK_TILE_K;

        // Load tiles from global to shared memory
        loadGlobalToShared(blockBaseK / 4, blockBaseM, BLOCK_TILE_VEC_K, BLOCK_TILE_M, K / 4, true);
        loadGlobalToShared(blockBaseVecN, blockBaseK, BLOCK_TILE_VEC_N, BLOCK_TILE_K, N / 4, false);
        __syncthreads();

        // Compute with shared memory using sub-tiles
#pragma unroll
        for (int iterTM = 0; iterTM < THREAD_TILE_M; iterTM += THREAD_SUBTILE_M) {
#pragma unroll
            for (int iterVecTN = 0; iterVecTN < THREAD_TILE_VEC_N; iterVecTN += THREAD_SUBTILE_VEC_N) {
                // Sub-tile computation
                float4 regA[THREAD_SUBTILE_VEC_K];
                float4 regB[THREAD_SUBTILE_K][THREAD_SUBTILE_VEC_N];

#pragma unroll
                for (int iterBK = 0; iterBK < BLOCK_TILE_K; iterBK += THREAD_TILE_K) {
#pragma unroll
                    for (int iterTK = 0; iterTK < THREAD_TILE_K; iterTK += THREAD_SUBTILE_K) {
                        // Load B registers
#pragma unroll
                        for (int iterSubTK = 0; iterSubTK < THREAD_SUBTILE_K; iterSubTK++) {
                            const int sharedCoordYB = iterBK + iterTK + iterSubTK;
#pragma unroll
                            for (int iterVecSubTN = 0; iterVecSubTN < THREAD_SUBTILE_VEC_N; iterVecSubTN++) {
                                const int sharedCoordVecXB = (iterVecTN + iterVecSubTN) * blockDim.x + tidx;
                                regB[iterSubTK][iterVecSubTN] = sharedB[sharedCoordYB][sharedCoordVecXB];
                            }
                        }

                        // Compute sub-tile
#pragma unroll
                        for (int iterSubTM = 0; iterSubTM < THREAD_SUBTILE_M; iterSubTM++) {
                            const int regCoordY = iterTM + iterSubTM;
                            const int sharedCoordYA = (iterTM + iterSubTM) * blockDim.y + tidy;

                            // Load A registers
#pragma unroll
                            for (int iterVecSubTK = 0; iterVecSubTK < THREAD_SUBTILE_VEC_K; iterVecSubTK++) {
                                const int sharedCoordVecXA = (iterBK + iterTK) / 4 + iterVecSubTK;
                                regA[iterVecSubTK] = sharedA[sharedCoordYA][sharedCoordVecXA];
                            }

                            // Outer product computation
#pragma unroll
                            for (int iterVecSubTN = 0; iterVecSubTN < THREAD_SUBTILE_VEC_N; iterVecSubTN++) {
                                const int regCoordVecX = iterVecTN + iterVecSubTN;
#pragma unroll
                                for (int iterVecSubTK = 0; iterVecSubTK < THREAD_SUBTILE_VEC_K; iterVecSubTK++) {
                                    const int regBaseYB = iterVecSubTK * 4;

                                    regAccumulator[regCoordY][regCoordVecX].x += regA[iterVecSubTK].x * regB[
                                        regBaseYB + 0][iterVecSubTN].x;
                                    regAccumulator[regCoordY][regCoordVecX].y += regA[iterVecSubTK].x * regB[
                                        regBaseYB + 0][iterVecSubTN].y;
                                    regAccumulator[regCoordY][regCoordVecX].z += regA[iterVecSubTK].x * regB[
                                        regBaseYB + 0][iterVecSubTN].z;
                                    regAccumulator[regCoordY][regCoordVecX].w += regA[iterVecSubTK].x * regB[
                                        regBaseYB + 0][iterVecSubTN].w;

                                    regAccumulator[regCoordY][regCoordVecX].x += regA[iterVecSubTK].y * regB[
                                        regBaseYB + 1][iterVecSubTN].x;
                                    regAccumulator[regCoordY][regCoordVecX].y += regA[iterVecSubTK].y * regB[
                                        regBaseYB + 1][iterVecSubTN].y;
                                    regAccumulator[regCoordY][regCoordVecX].z += regA[iterVecSubTK].y * regB[
                                        regBaseYB + 1][iterVecSubTN].z;
                                    regAccumulator[regCoordY][regCoordVecX].w += regA[iterVecSubTK].y * regB[
                                        regBaseYB + 1][iterVecSubTN].w;

                                    regAccumulator[regCoordY][regCoordVecX].x += regA[iterVecSubTK].z * regB[
                                        regBaseYB + 2][iterVecSubTN].x;
                                    regAccumulator[regCoordY][regCoordVecX].y += regA[iterVecSubTK].z * regB[
                                        regBaseYB + 2][iterVecSubTN].y;
                                    regAccumulator[regCoordY][regCoordVecX].z += regA[iterVecSubTK].z * regB[
                                        regBaseYB + 2][iterVecSubTN].z;
                                    regAccumulator[regCoordY][regCoordVecX].w += regA[iterVecSubTK].z * regB[
                                        regBaseYB + 2][iterVecSubTN].w;

                                    regAccumulator[regCoordY][regCoordVecX].x += regA[iterVecSubTK].w * regB[
                                        regBaseYB + 3][iterVecSubTN].x;
                                    regAccumulator[regCoordY][regCoordVecX].y += regA[iterVecSubTK].w * regB[
                                        regBaseYB + 3][iterVecSubTN].y;
                                    regAccumulator[regCoordY][regCoordVecX].z += regA[iterVecSubTK].w * regB[
                                        regBaseYB + 3][iterVecSubTN].z;
                                    regAccumulator[regCoordY][regCoordVecX].w += regA[iterVecSubTK].w * regB[
                                        regBaseYB + 3][iterVecSubTN].w;
                                }
                            }
                        }
                    }
                }
            }
        }
        __syncthreads();
    }

    // Store results to global memory
    const int globalBaseY = bidy * BLOCK_TILE_M + tidy;
    const int globalBaseVecX = bidx * BLOCK_TILE_VEC_N + tidx;

#pragma unroll
    for (int tm = 0; tm < THREAD_TILE_M; tm++) {
#pragma unroll
        for (int tn = 0; tn < THREAD_TILE_VEC_N; tn++) {
            const int globalCoordY = globalBaseY + tm * blockDim.y;
            const int globalCoordVecX = globalBaseVecX + tn * blockDim.x;
            const int dstIdx = globalCoordY * (N / 4) + globalCoordVecX;
            dstMat[dstIdx] = regAccumulator[tm][tn];
        }
    }
}


void runCublas(hipblasHandle_t handle, int M, int N, int K, float alpha,
               float* A, float* B, float beta, float* C) {
    // cuBLAS uses column-major order. So we change the order of our row-major A &
    // B, since (B^T*A^T)^T = (A*B)
    // This runs cuBLAS in full fp32 mode
    hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, B, HIP_R_32F,
                 N, A, HIP_R_32F, K, &beta, C, HIP_R_32F, N, HIPBLAS_COMPUTE_32F,
                 CUBLAS_GEMM_DEFAULT_TENSOR_OP);
}


void runMySgemm(int M, int N, int K, float alpha, float* A, float* B, float beta, float* C) {
    constexpr uint32_t BM = 128;
    constexpr uint32_t BN = 64;
    constexpr uint32_t BK = 16;
    constexpr uint32_t TM = 8;
    constexpr uint32_t TN = 8;
    constexpr uint32_t TK = 4;
    constexpr uint32_t STM = 4;
    constexpr uint32_t STN = 8;
    constexpr uint32_t STK = 4;
    dim3 gridDim(N / BN, M / BM);
    dim3 blockDim(BN / TN, BM / TM);
    sgemmSubTile<BM, BN, BK, TM, TN, TK, STM, STN, STK> <<<gridDim, blockDim>>>(
        M, N, K, (float4*)A, (float4*)B, (float4*)C);
}

void randomizeMat(float* mat, int N) {
    std::mt19937 rdEngine;
    rdEngine.seed(37);
    std::uniform_real_distribution dist(0.0f, 1.0f);
    for (int i = 0; i < N; i++) {
        mat[i] = dist(rdEngine);
    }
}

bool verifyMat(float* matRef, float* matOut, int N) {
    double diff = 0.0;
    int i;
    for (i = 0; i < N; i++) {
        diff = std::fabs(matRef[i] - matOut[i]);
        if (isnan(diff) || diff > 0.01) {
            printf("expect %5.2f, get %5.2f at %d\n", matRef[i], matOut[i], i);
            return false;
        }
    }
    return true;
}

int main(int argc, char** argv) {
    hipblasHandle_t handle;
    if (hipblasCreate(&handle)) {
        std::cerr << "Create cublas handle error." << std::endl;
        exit(EXIT_FAILURE);
    }

    hipEvent_t evBegin, evEnd;
    hipEventCreate(&evBegin);
    hipEventCreate(&evEnd);

    // cuBLAS FLOPs ceiling is reached at 8192
    std::array SIZES{2048, 3072, 4096};

    int64_t m, n, k;
    int64_t max_size = SIZES.back();
    std::cout << "Max size: " << max_size << std::endl;

    float alpha = 1.0, beta = 0.0; // GEMM input parameters, C=α*AB+β*C

    float *A = nullptr, *B = nullptr, *C = nullptr, *C_ref = nullptr;
    float *dA = nullptr, *dB = nullptr, *dC = nullptr, *dC_ref = nullptr;

    A = (float*)malloc(sizeof(float) * max_size * max_size);
    B = (float*)malloc(sizeof(float) * max_size * max_size);
    C = (float*)malloc(sizeof(float) * max_size * max_size);
    C_ref = (float*)malloc(sizeof(float) * max_size * max_size);

    randomizeMat(A, max_size * max_size);
    randomizeMat(B, max_size * max_size);
    randomizeMat(C, max_size * max_size);

    CHECK_CUDA(hipMalloc((void**)&dA, sizeof(float) * max_size * max_size));
    CHECK_CUDA(hipMalloc((void**)&dB, sizeof(float) * max_size * max_size));
    CHECK_CUDA(hipMalloc((void**)&dC, sizeof(float) * max_size * max_size));
    CHECK_CUDA(hipMalloc((void**)&dC_ref, sizeof(float) * max_size * max_size));

    CHECK_CUDA(hipMemcpy(dA, A, sizeof(float) * max_size * max_size, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(dB, B, sizeof(float) * max_size * max_size, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(dC, C, sizeof(float) * max_size * max_size, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(dC_ref, C, sizeof(float) * max_size * max_size, hipMemcpyHostToDevice));

    int64_t PERF_TIMES = 1;
    for (int size : SIZES) {
        m = n = k = size;
        runCublas(handle, m, n, k, alpha, dA, dB, beta, dC_ref);
        runMySgemm(m, n, k, alpha, dA, dB, beta, dC);
        CHECK_CUDA(hipDeviceSynchronize());
        CHECK_CUDA(hipGetLastError()); // Check for async errors during kernel run
        hipMemcpy(C, dC, sizeof(float) * m * n, hipMemcpyDeviceToHost);
        hipMemcpy(C_ref, dC_ref, sizeof(float) * m * n, hipMemcpyDeviceToHost);

        if (!verifyMat(C_ref, C, m * n)) {
            std::cout << "verification failed" << std::endl;
            exit(EXIT_FAILURE);
        }

        std::vector<float> elapsedTimes;
        for (int j = 0; j < PERF_TIMES; j++) {
            hipEventRecord(evBegin);
            runMySgemm(m, n, k, alpha, dA, dB, beta, dC);
            hipEventRecord(evEnd);
            hipEventSynchronize(evBegin);
            hipEventSynchronize(evEnd);
            float elapsedTime;
            hipEventElapsedTime(&elapsedTime, evBegin, evEnd);
            elapsedTimes.push_back(elapsedTime);
        }

        const auto [meanTime, stdTime] = meanStd(elapsedTimes);
        const float macs = (float)m * n * k * 2;
        const float meanTflops = macs / meanTime / 1e9;

        std::cout << "=============================" << std::endl;
        std::cout << "Size: " << size << std::endl;
        std::cout << "Performance: " << meanTflops << " tflops" << std::endl;
    }

    // Free up CPU and GPU space
    free(A);
    free(B);
    free(C);
    free(C_ref);
    hipFree(dA);
    hipFree(dB);
    hipFree(dC);
    hipFree(dC_ref);
    hipblasDestroy(handle);

    return 0;
};