﻿#include <hipblas.h>
#include <hip/hip_runtime.h>

#include <array>
#include <iostream>
#include <vector>

#include "../cud_helper.hpp"

int main() {
    constexpr std::array SIZES{2048, 3072, 4096};
    constexpr int HEATUP_TIMES = 1;
    constexpr int PERF_TIMES = 3;

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    hipEvent_t evBegin, evEnd;
    hipEventCreate(&evBegin);
    hipEventCreate(&evEnd);

    for (const int size : SIZES) {
        const int M = size;
        const int N = size;
        const int K = size;

        float *deviceA, *deviceB, *deviceC;
        hipMalloc(&deviceA, M * K * sizeof(float));
        hipMalloc(&deviceB, K * N * sizeof(float));
        hipMalloc(&deviceC, M * N * sizeof(float));

        std::vector hostA(M * K, 1.0f);
        std::vector hostB(K * N, 2.0f);

        hipMemcpy(deviceA, hostA.data(), M * K * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(deviceB, hostB.data(), K * N * sizeof(float), hipMemcpyHostToDevice);

        const float alpha = 1.0f;
        const float beta = 0.0f;

        constexpr hipblasGemmAlgo_t algo = CUBLAS_GEMM_ALGO0;

        for (int i = 0; i < HEATUP_TIMES; i++) {
            hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, deviceB, HIP_R_32F,
                         N, deviceA, HIP_R_32F, K, &beta, deviceC, HIP_R_32F, N, HIPBLAS_COMPUTE_32F,
                         algo);
            hipDeviceSynchronize();
        }

        std::vector<float> elapsedTimes;
        float elapsedTime = 0;
        for (int i = 0; i < PERF_TIMES; i++) {
            hipEventRecord(evBegin);
            hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, deviceB, HIP_R_32F,
                         N, deviceA, HIP_R_32F, K, &beta, deviceC, HIP_R_32F, N, HIPBLAS_COMPUTE_32F,
                         algo);
            hipEventRecord(evEnd);
            hipEventSynchronize(evEnd);
            hipEventElapsedTime(&elapsedTime, evBegin, evEnd);
            elapsedTimes.push_back(elapsedTime);
        }

        const auto [meanTime, stdTime] = meanStd(elapsedTimes);
        const float macs = (float)M * N * K * 2;
        const float meanTflops = macs / meanTime / 1e9;

        std::cout << "=============================" << std::endl;
        std::cout << "Size: " << size << std::endl;
        std::cout << "Performance: " << meanTflops << " tflops" << std::endl;

        hipFree(deviceA);
        hipFree(deviceB);
        hipFree(deviceC);
    }

    hipEventDestroy(evBegin);
    hipEventDestroy(evEnd);
    hipblasDestroy(handle);
}