﻿#include <hipblaslt.h>
#include <hip/hip_runtime.h>

#include <array>
#include <iostream>
#include <vector>

#include "../cud_helper.hpp"

int main() {
    constexpr std::array SIZES{1024, 2048, 3072, 4096, 5120, 6144, 7168, 8192, 10240};
    constexpr int HEATUP_TIMES = 1;
    constexpr int PERF_TIMES = 3;

    hipblasLtHandle_t ltHandle;
    hipblasLtCreate(&ltHandle);

    hipblasLtMatmulDesc_t operationDesc;
    hipblasLtMatmulDescCreate(&operationDesc, HIPBLAS_COMPUTE_32F, HIP_R_32F);

    hipblasOperation_t transA = HIPBLAS_OP_N;
    hipblasOperation_t transB = HIPBLAS_OP_N;
    hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &transA, sizeof(transA));
    hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &transB, sizeof(transB));

    hipblasLtMatmulPreference_t preference;
    hipblasLtMatmulPreferenceCreate(&preference);

    hipEvent_t evBegin, evEnd;
    hipEventCreate(&evBegin);
    hipEventCreate(&evEnd);

    for (const int size : SIZES) {
        const int M = size;
        const int N = size;
        const int K = size;

        float *deviceA, *deviceB, *deviceC;
        hipMalloc(&deviceA, M * K * sizeof(float));
        hipMalloc(&deviceB, K * N * sizeof(float));
        hipMalloc(&deviceC, M * N * sizeof(float));

        std::vector hostA(M * K, 1.0f);
        std::vector hostB(K * N, 2.0f);

        hipMemcpy(deviceA, hostA.data(), M * K * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(deviceB, hostB.data(), K * N * sizeof(float), hipMemcpyHostToDevice);

        const float alpha = 1.0f;
        const float beta = 0.0f;

        hipblasLtMatrixLayout_t layoutA, layoutB, layoutC;
        hipblasLtMatrixLayoutCreate(&layoutA, HIP_R_32F, M, K, M);
        hipblasLtMatrixLayoutCreate(&layoutB, HIP_R_32F, K, N, K);
        hipblasLtMatrixLayoutCreate(&layoutC, HIP_R_32F, M, N, M);

        constexpr int expectAlgoCount = 1;
        std::array<hipblasLtMatmulHeuristicResult_t, expectAlgoCount> heuristicResult;
        int returnedResults = 0;

        hipblasLtMatmulAlgoGetHeuristic(ltHandle, operationDesc, layoutA, layoutB, layoutC, layoutC, preference,
                                       expectAlgoCount,
                                       heuristicResult.data(), &returnedResults);

        if (returnedResults == 0) {
            std::cerr << "No suitable SIMT algorithm found!" << std::endl;
            exit(EXIT_FAILURE);
        }

        for (int i = 0; i < HEATUP_TIMES; i++) {
            hipblasLtMatmul(ltHandle, operationDesc, &alpha, deviceB, layoutB, deviceA, layoutA, &beta, deviceC, layoutC,
                           deviceC, layoutC,
                           &heuristicResult[0].algo, nullptr, 0, nullptr);
            hipDeviceSynchronize();
        }

        std::vector<float> elapsedTimes;
        float elapsedTime = 0;
        for (int i = 0; i < PERF_TIMES; i++) {
            hipEventRecord(evBegin);
            hipblasLtMatmul(ltHandle, operationDesc, &alpha, deviceB, layoutB, deviceA, layoutA, &beta, deviceC, layoutC,
                           deviceC, layoutC,
                           &heuristicResult[0].algo, nullptr, 0, nullptr);
            hipEventRecord(evEnd);
            hipEventSynchronize(evEnd);
            hipEventElapsedTime(&elapsedTime, evBegin, evEnd);
            elapsedTimes.push_back(elapsedTime);
        }

        const auto [meanTime, stdTime] = meanStd(elapsedTimes);
        const float macs = (float)M * N * K * 2;
        const float meanTflops = macs / meanTime / 1e9;

        // std::cout << "=============================" << std::endl;
        // std::cout << "Size: " << size << std::endl;
        // std::cout << "Performance: " << meanTflops << " tflops" << std::endl;
        std::cout << meanTflops << std::endl;

        hipblasLtMatrixLayoutDestroy(layoutA);
        hipblasLtMatrixLayoutDestroy(layoutB);
        hipblasLtMatrixLayoutDestroy(layoutC);
        hipFree(deviceA);
        hipFree(deviceB);
        hipFree(deviceC);
    }

    hipEventDestroy(evBegin);
    hipEventDestroy(evEnd);
    hipblasLtMatmulPreferenceDestroy(preference);
    hipblasLtMatmulDescDestroy(operationDesc);
    hipblasLtDestroy(ltHandle);
}